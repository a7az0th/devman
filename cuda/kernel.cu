﻿#include "hip/hip_runtime.h"
#include "../utils.h"


#ifdef __HIPCC__

extern "C"
KERNEL void fillmatrix(int* m0, int* m1, int* res, int size) {
	const int x = threadIdx.x % size;
	const int y = threadIdx.x / size;

	Matrix<int> mm0;
	mm0.init(m0, size, size);

	Matrix<int> mm1;
	mm1.init(m1, size, size);

	Matrix<int> r;
	r.init(res, size, size);


	int& result = r.getRow(y)[x];
	for (int p = 0; p < 10000; p++) {
		result = 0;
		for (int i=0; i < size; i++) {
			result += mm0.getRow(y)[i] * mm1.getRow(i)[x];
		}
	}
}


extern "C"
KERNEL void dummy() {
}

#endif