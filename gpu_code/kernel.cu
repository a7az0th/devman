﻿#include "hip/hip_runtime.h"
#include "utils.h"


extern "C"
KERNEL void dummy(int* res) {
	if (getGlobalID(0) == 0) {
		res[0] = 1337;
	}
}
