﻿#include "hip/hip_runtime.h"
#include "utils.h"


extern "C"
KERNEL void dummy(float *C, float *A, float *B) {
    
	const int idx = getGlobalID(0);

	const int x = idx % WIDTH;
	const int y = idx / WIDTH;

    float sum = 0.0;

    for (int k=0; k<WIDTH; k++) {
        sum += A[y*WIDTH+k] * B[k*WIDTH+x];
    }

    C[y*WIDTH+x] = sum;
}
