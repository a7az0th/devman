#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "utils.h"


extern "C"
KERNEL void dummyGlobal(float *C, float *A, float *B) {
    
	const int idx = getGlobalID(0);

	const int x = idx % WIDTH;
	const int y = idx / WIDTH;

    float sum;

	for (int i = 0; i < 3*10000000; i++) {
		sum = 0.0f;
		for (int k=0; k<WIDTH; k++) {
		    sum += A[y*WIDTH+k] * B[k*WIDTH+x];
		}
	}

    C[y*WIDTH+x] = sum;
}


extern "C"
KERNEL void dummyShared(float *C, float *A, float *B) {
    
	const int idx = getGlobalID(0);

	const int x = idx % WIDTH;
	const int y = idx / WIDTH;

	__shared__ float localA[WIDTH][WIDTH];
	__shared__ float localB[WIDTH][WIDTH];

	localA[x][y] = A[y*WIDTH+x];
	localB[x][y] = B[y*WIDTH+x];
	__syncthreads();

    float sum;

	for (int i = 0; i < 3*10000000; i++) {
		sum = 0.0f;
		for (int k=0; k<WIDTH; k++) {
		    sum += localA[y][k] * localB[k][x];
		}
	}

    C[y*WIDTH+x] = sum;
}
