#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "utils.h"

const int N = 1 << 20;

extern "C"
KERNEL void kernel(float *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = sqrt(pow(3.14159,i));
    }
}

extern "C"
KERNEL void dummyGlobal(float *C, float *A, float *B) {
    
	const int idx = getGlobalID(0);

	const int x = idx % WIDTH;
	const int y = idx / WIDTH;

    float sum;

	for (int i = 0; i < 3*10000000; i++) {
		sum = 0.0f;
		for (int k=0; k<WIDTH; k++) {
		    sum += A[y*WIDTH+k] * B[k*WIDTH+x];
		}
	}

    C[y*WIDTH+x] = sum;
}


extern "C"
KERNEL void dummyShared(float *C, float *A, float *B) {
    
	const int idx = getGlobalID(0);

	const int x = idx % WIDTH;
	const int y = idx / WIDTH;

	__shared__ float localA[WIDTH][WIDTH];
	__shared__ float localB[WIDTH][WIDTH];

	localA[x][y] = A[y*WIDTH+x];
	localB[x][y] = B[y*WIDTH+x];
	__syncthreads();

    float sum;

	for (int i = 0; i < 3*10000000; i++) {
		sum = 0.0f;
		for (int k=0; k<WIDTH; k++) {
		    sum += localA[y][k] * localB[k][x];
		}
	}

    C[y*WIDTH+x] = sum;
}
