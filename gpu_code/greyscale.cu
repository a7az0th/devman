#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "utils.h"


extern "C"
KERNEL void greyscale(float3 *buffIn, float3* buffOut) {
	const int i = threadIdx.x + blockIdx.x * blockDim.x;
	//for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
	const float3 col = buffIn[i];
	const float R = col.x;
	const float G = col.y;
	const float B = col.z;
	const float luminance = 0.2126f*R + 0.7152f*G + 0.0722f*B;
	buffOut[i] = make_float3(luminance, luminance, luminance);
}
